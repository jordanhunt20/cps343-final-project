#include "hip/hip_runtime.h"
/*
 * $Smake: nvcc -O2 -o %F %f wtime.c -I ../include -lhdf5
 *
 * Matrix-matrix product
 */

#include <hip/hip_runtime.h>
#include <hdf5.h>
#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <cblas.h>
#include <cmath>
#include <iostream>
#include "wtime.h"



/// Check return values from HDF5 routines
#define CHKERR(status,name) if ( status < 0 ) \
     fprintf( stderr, "Error: nonzero status (%d) in %s\n", status, name )

// Compute index into single linear array for matrix element (i,j)
#define IDX(i,j,stride) ((i)*(stride)+(j)) // row major (c/c++ ordering)

//----------------------------------------------------------------------------
// Display matrix values on standard output

void dumpMatrix(
    double* a,      // in  - address of matrix data
    int rows,       // in  - number of rows in matrix
    int cols,       // in  - number of cols in matrix
    int stride      // in  - row length in memory (assuming C/C++ storage)
    )
{
    for ( int i = 0; i < rows; i++ )
    {
        for ( int j = 0; j < cols; j++ )
        {
            printf( " %8.2f", a[IDX(i,j,stride)] );
        }
        printf( "\n" );
    }
    printf( "\n" );
    fflush( stdout );
}

//----------------------------------------------------------------------------//
//----------------------------------------------------------------------------//
//					MATRIX OPERATIONS										  //
//----------------------------------------------------------------------------//
//----------------------------------------------------------------------------//

/*			- 					tested
 * Form vector product result = Ab
 * @param1 double* vector result
 * @param2 double* matrix to multiply
 * @param3 double* vector to multiply
 * @param4 int number of rows in matrix
 */
 __global__ void mat_vec_mult(double* result, double* A, double* b, int numRows)
 {
     int row = blockIdx.y * blockDim.y + threadIdx.y;
     int col = blockIdx.x * blockDim.x + threadIdx.x;
     double sum = 0.0;
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if (tid < numRows * numRows) {
         for (int i = 0; i < numRows; i++) {
             sum += A[i * numRows + tid] * b[i];
         }
         result[row * numRows + col] = sum;
     }
 }

 /*
 * Form int product c = a^T * b  -  tested
*/
void vec_vec_mult( double* c, double* a, int rows, double* b)
{
	double sum = 0.0;
    int i;
	for (i = 0; i < rows; i++)
	{
		sum += a[i] * b[i];
	}
    *c = sum + 0.0;
}

/*
 * Form vector quotient c = a/||a||  -   tested
*/
void normalize( double* c, double* a, int rows )
{
	double magnitude = 0.0;
    for (int i = 0; i < rows; i++)
	{
		magnitude += pow(a[i], 2.0);
	}
    for (int i = 0; i < rows; i++)
	{
		c[i] = a[i] / sqrt(magnitude);
    }
}

//----------------------------------------------------------------------------
//----------------------------------------------------------------------------


int main( int argc, char* argv[] )
{
	double tolerance = pow(10, -6.0); // default value
    long numIterations = 500; // default value
    long blockSize = 16; // default value

    double* a;             // pointer to matrix data
    hid_t file_id;         // HDF5 id for file
    hid_t dataspace_id;    // HDF5 id for dataspace in file
    hid_t dataset_id;      // HDF5 id for dataset in file
    hid_t memspace_id;     // HDF5 id for dataset in memory
    hsize_t* dims;         // matrix dimensions
    herr_t status;         // HDF5 return code
    int ndim;              // number of dimensions in HDF5 dataset

	// Process command line
	int c;

    const char* filename = argv[1];
    const char* path = "/A/value";

    bool quiet = false;

    // check for switches
    while ( ( c = getopt( argc, argv, "e:m:s:q" ) ) != -1 )
	{
	    switch( c )
		{
		case 'e':
		    tolerance = atof(optarg);
            if (tolerance <= 0)
            {
                fprintf(stderr, "tolerance must be positive\n");
                fprintf(stderr, "got: %f\n", tolerance);
                exit(EXIT_FAILURE);
            }
		    break;
		case 'm':
		    numIterations = atol(optarg);
	    	if (numIterations <= 0)
			{
			    fprintf(stderr, "number of iterations must be positive\n");
			    fprintf(stderr, "got: %ld\n", numIterations);
			    exit(EXIT_FAILURE);
			}
		    break;
        case 's':
            blockSize = atol( optarg );
            if ( blockSize <= 0 )
            {
                fprintf(stderr, "block size must be positive\n");
                fprintf(stderr, "got: %ld\n", blockSize);
                exit(EXIT_FAILURE);
            }
            break;
        case 'q':
            quiet = true;
            break;
		default:
		    fprintf(stderr, "default usage: %s [-e tolerance, -m numIterations -s blockSize, -q]\n", argv[0]);
		    return EXIT_FAILURE;
		}
	}

	//----------------------------------------------------------------------------//
	//----------------------------------------------------------------------------//
	//					READ MATRIX     										  //
	//----------------------------------------------------------------------------//
	//----------------------------------------------------------------------------//

	double startTime = wtime();

    // Open existing HDF5 file
    file_id = H5Fopen( filename, H5F_ACC_RDONLY, H5P_DEFAULT );
    if ( file_id < 0 ) exit( EXIT_FAILURE );

    // Open dataset in file
    dataset_id = H5Dopen( file_id, path, H5P_DEFAULT );
    if ( dataset_id < 0 ) exit( EXIT_FAILURE );

    // Determine dataset parameters
    dataspace_id = H5Dget_space( dataset_id );
    ndim = H5Sget_simple_extent_ndims( dataspace_id );
    dims = new hsize_t [ndim];

    // Get dimensions for dataset
    ndim = H5Sget_simple_extent_dims( dataspace_id, dims, NULL );
    if ( ndim != 2 )
    {
        fprintf( stderr, "Expected dataspace to be 2-dimensional " );
        fprintf( stderr, "but it appears to be %d-dimensional\n", ndim );
        exit( EXIT_FAILURE );
    }

    // Create memory dataspace
    memspace_id = H5Screate_simple(ndim, dims, NULL);
    if (memspace_id < 0) exit(EXIT_FAILURE);

    // Allocate memory for matrix and read data from file
    a = new double [dims[0] * dims[0]];
    status = H5Dread(dataset_id, H5T_NATIVE_DOUBLE, memspace_id,
                      dataspace_id, H5P_DEFAULT, a);
    CHKERR(status, "H5Dread()");

    // Close all remaining HDF5 objects
    CHKERR(H5Sclose(memspace_id), "H5Sclose()");
    CHKERR(H5Dclose(dataset_id), "H5Dclose()");
    CHKERR(H5Sclose(dataspace_id), "H5Sclose()");
    CHKERR(H5Fclose(file_id), "H5Fclose()");

	double endTime = wtime();
	double readTime = endTime - startTime;

    startTime = endTime;

//----------------------------------------------------------------------------
//----------------------------------------------------------------------------

    // Power Method Algorithm
	int cols = (int) dims[0];
    double x [cols]; // corresponding normalized eigenvector
    double y [cols]; // placeholder

    //initial eigenvector estimate (using y as a placeholder)
    for ( int i = 0; i < cols; i++) y[i] = 1.0;

    //normalize x (based on placeholder y)
    normalize(x, y, cols);

    // initialized to any value
    double lambda = 0.0;

    // make sure |lambda-lambda_0| > tolerance
    double lambda_0 = lambda + 2 * tolerance;

    // determine matrix size in bytes
    size_t matrix_size = cols * cols * sizeof(double);

    // determine vector size in bytes
    size_t vector_size = cols * sizeof(double);

    // set block size and number of blocks
    dim3 block_size(blockSize, blockSize);
  	dim3 num_blocks((cols - 1 + block_size.x) / block_size.x,
                   (cols - 1 + block_size.y) / block_size.y);

    // declare pointers to matrix and vectors in device memory and allocate memory
    double *a_d, *x_d, *y_d;
    hipMalloc((void**) &a_d, matrix_size); // matrix
    hipMalloc((void**) &x_d, vector_size); // eigenvalue
    hipMalloc((void**) &y_d, vector_size); // placeholder

    // set a_d to matrix a, and copy it to device
    hipMemcpy(a_d, a, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(x_d, x, vector_size, hipMemcpyHostToDevice);

    long k = 0;
	while ((std::abs(lambda - lambda_0) >= tolerance) && k <= numIterations)
	{
         // compute next eigenvector estimate on device
        mat_vec_mult<<< num_blocks, block_size >>>(y_d, a_d, x_d, cols);

        // retrieve result from device and store on host
        hipMemcpy(y, y_d, vector_size, hipMemcpyDeviceToHost);
        // for(int i = 0; i < cols/2; i++)
        //     std::cout << y[i] << std::endl;
		lambda_0 = lambda; 						      	// previous eigenvalue estimate
 		vec_vec_mult(&lambda, x, cols, y);    			// compute new estimate
		normalize(x, y, cols );						// normalize eigenvector estimate

        // copy x from host to device
        hipMemcpy(x_d, x, vector_size, hipMemcpyHostToDevice);
		k++;
	}

    double executionTime = wtime() - startTime;

    if (quiet) {
        printf("\n%f %ld %f\n", lambda, k, readTime + executionTime);
    } else {
        printf("\nDominant Eigenvalue: %f\nRead Time: %f\nNumber Of Iterations: %ld\nExecution Time: %f\n", lambda, readTime, k, executionTime);
        printf("Number of Processes: %d\nTotal Time: %f\nNumber of Processes * Total Time: %f\nTime Per Loop: %f\n\n", 1, readTime + executionTime, readTime + executionTime, executionTime / (k + 0.0));
    }
//----------------------------------------------------------------------------
//----------------------------------------------------------------------------
    // Clean up and quit
    delete [] a;
    delete [] dims;
}
